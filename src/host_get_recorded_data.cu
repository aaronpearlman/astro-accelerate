#include <stdio.h>
#include <hip/hip_runtime.h>

/* Again this funtion uses a pointer to the file pointer so that it can update the position of the file pointer :)
 * Note the brakets surrounding the input_buffer, these are needed due to operator percidence...
 * (*input_buffer_odd)[(c*(*nsamp))  + total_data] = (float)temp_buffer[c];
 */

void get_recorded_data(FILE **fp, int nsamp, int nchans, int nbits, unsigned short **input_buffer, size_t *inputsize)
{

	int c;

	unsigned long int total_data;

	//{{{ Load in the raw data from the input file and transpose
	if (nbits == 32)
	{
		// Allocate a tempory buffer to store a line of frequency data
		float *temp_buffer = (float *) malloc(nchans * sizeof(float));

		// Allocate floats to hold the mimimum and maximum value in the input data
		float max = -10000000000;
		float min = 10000000000;

		// Allocate a variable to hold the file pointer position
		fpos_t file_loc;

		// Set the file pointer position
		fgetpos(*fp, &file_loc);
		
		// Find the minimum and maximum values in the input file.
		while (!feof(*fp))
		{
		  if (fread(temp_buffer, sizeof(float), nchans, *fp) != (size_t)nchans)
				break;
			for (c = 0; c < nchans; c++)
			{
				if(temp_buffer[c] > max) max = temp_buffer[c];
				if(temp_buffer[c] < min) min = temp_buffer[c];
			}
		}

		// Calculate the bin size in a distribution of unsigned shorts for the input data.
		float bin = (max - min) / 65535.0f;

		printf("\n Conversion Parameters: %f\t%f\t%f", min, max, bin);

		// Move the file pointer back to the end of the header
		fsetpos(*fp, &file_loc);

		// Read in the data, scale to an unsigned short range, transpose it and store it in the input buffer
		total_data = 0;
		while (!feof(*fp))
		{
		  if (fread(temp_buffer, sizeof(float), nchans, *fp) != (size_t)nchans)
				break;
			for (c = 0; c < nchans; c++)
			{
				( *input_buffer )[c + total_data * ( nchans )] = (unsigned short) ((temp_buffer[c]-min)/bin);
			}
			total_data++;
		}
		free(temp_buffer);
	} 
	else if (nbits == 16)
	{

		// Allocate a tempory buffer to store a line of frequency data
		unsigned short *temp_buffer = (unsigned short *) malloc(nchans * sizeof(unsigned short));

		// Read in the data, transpose it and store it in the input buffer
		total_data = 0;
		while (!feof(*fp))
		{

		  if (fread(temp_buffer, sizeof(unsigned short), nchans, *fp) != (size_t)nchans)
				break;
			for (c = 0; c < nchans; c++)
			{
				( *input_buffer )[c + total_data * ( nchans )] = (unsigned short) temp_buffer[c];
			}
			total_data++;
		}
		free(temp_buffer);
	}
	else if (nbits == 8)
	{

		// Allocate a tempory buffer to store a line of frequency data
		unsigned char *temp_buffer = (unsigned char *) malloc(nchans * sizeof(unsigned char));

		// Read in the data, transpose it and store it in the input buffer
		total_data = 0;
		while (!feof(*fp))
		{

		  if (fread(temp_buffer, sizeof(unsigned char), nchans, *fp) != (size_t)nchans)
				break;
			for (c = 0; c < nchans; c++)
			{
				( *input_buffer )[c + total_data * ( nchans )] = (unsigned short) temp_buffer[c];
			}
			total_data++;
		}
		free(temp_buffer);
	}
	else if (nbits == 4)
	{
		// Allocate a temporary buffer to store a line of frequency data
		// each byte stores 2 frequency data
		int nb_bytes = nchans/2;
		unsigned char *temp_buffer = (unsigned char *) malloc(nb_bytes * sizeof(unsigned char));
		// Read in the data, transpose it and store it in the input buffer
		total_data = 0;
		// 00001111
		char mask = 0x0f;
		while (!feof(*fp))
		{
		  if (fread(temp_buffer, sizeof(unsigned char), nb_bytes, *fp) != (size_t)nb_bytes)
				break;
			for (c = 0; c < nb_bytes; c++)
			{
				// (n >> a) & ( (1 << a) - 1) -> right shift by 'a' bits, then keep the last 'b' bits
				// Here, we right shift 4 bits and keep the last 4 bits
				( *input_buffer )[ (c*2) + total_data * ( nchans )]     = (unsigned short)( (temp_buffer[c] >> 4) & mask );
				// n & ( (1 << a ) - 1)
				// Here we keep the last 4 bits
				( *input_buffer )[ (c*2) + 1 + total_data * ( nchans )] = (unsigned short)( temp_buffer[c] & mask );
			}
			total_data++;
		}
		free(temp_buffer);
	}
	else if (nbits == 2)
	{
		// Allocate a temporary buffer to store a line of frequency data
		// each byte stores 4 frequency data
		int nb_bytes = nchans/4;
		unsigned char *temp_buffer = (unsigned char *) malloc(nb_bytes * sizeof(unsigned char));
		// Read in the data, transpose it and store it in the input buffer
		total_data = 0;
		// 00001111
		char mask = 0x03;
		while (!feof(*fp))
		{
		  if (fread(temp_buffer, sizeof(unsigned char), nb_bytes, *fp) != (size_t)nb_bytes)
				break;
			for (c = 0; c < nb_bytes; c++)
			{
				( *input_buffer )[ (c*4) + total_data * ( nchans )]     = (unsigned short)( (temp_buffer[c] >> 6) & mask );
				( *input_buffer )[ (c*4) + 1 + total_data * ( nchans )] = (unsigned short)( (temp_buffer[c] >> 4) & mask );
				( *input_buffer )[ (c*4) + 2 + total_data * ( nchans )] = (unsigned short)( (temp_buffer[c] >> 2) & mask );
				( *input_buffer )[ (c*4) + 3 + total_data * ( nchans )] = (unsigned short)( temp_buffer[c] & mask );
			}
			total_data++;
		}
		free(temp_buffer);
	}
	else if (nbits == 1)
	{
		// each byte stores 8 frequency data
		int nb_bytes = nchans/8;

		// Allocate a temporary buffer to store a line of frequency data
		unsigned char *temp_buffer = (unsigned char *) malloc(nb_bytes * sizeof(unsigned char));

		// Read in the data, transpose it and store it in the input buffer
		total_data = 0;

		while (!feof(*fp))
		{
		  if (fread(temp_buffer, sizeof(unsigned char), nb_bytes, *fp) != (size_t)nb_bytes)
				break;

			for (c = 0; c < nb_bytes; c++)
			{
				for(int i=0; i<8; i++){
					unsigned char mask =  1 << i;
					unsigned char masked_char = temp_buffer[c] & mask;
					unsigned char bit = masked_char >> i;
					( *input_buffer )[ (c*8) + i + total_data * ( nchans )] = (unsigned short)bit;
				}
			}
			total_data++;
		}
		free(temp_buffer);
	}
	else
	{
		printf("\n\n=============================== ERROR ====================================\n");
		printf(    " This is a SKA prototype code and only runs with 4, 8, 16 and 32 bit data\n");
		printf(  "\n==========================================================================\n");
		exit(0);
	}

	//}}}
}
