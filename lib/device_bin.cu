//#include <hip/hip_runtime_api.h>
// #include <omp.h>
#include <time.h>
#include <stdio.h>
#include "headers/params.h"
#include "device_binning_kernel.cu"
#include "timer.h"

//extern "C" void bin_gpu(float *bin_buffer, float *input_buffer, int nchans, int nsamp);

void bin_gpu(unsigned short *d_input, float *d_output, int nchans, int nsamp)
{

	GpuTimer timer;
	int divisions_in_t = BINDIVINT;
	int divisions_in_f = BINDIVINF;
	int num_blocks_t = (int) ( ( nsamp + 1 ) / ( 2*divisions_in_t ) );
	int num_blocks_f = nchans / divisions_in_f;
	
	//printf("\ndivisions_in_t:%d\tdivisions_in_f:%d",divisions_in_t, divisions_in_f);
	//printf("\nnum_blocks_t:%d\tnum_blocks_f:%d",num_blocks_t, num_blocks_f);
	//printf("\nDIVISOR_t: %f\t DIVISOR_f: %f\n", (float)( nsamp + 1 )/( 2*divisions_in_t ), (float) nchans/divisions_in_f);

	dim3 threads_per_block(divisions_in_t, divisions_in_f);
	dim3 num_blocks(num_blocks_t, num_blocks_f);
	
	//hipFuncSetCacheConfig(reinterpret_cast<const void*>(bin), hipFuncCachePreferL1);
	timer.Start();

	bin<<<num_blocks, threads_per_block>>>(d_input, d_output, nsamp);
	//	getLastCudaError("Kernel execution failed");

	int swap_divisions_in_t = CT;
	int swap_divisions_in_f = CF;
	int swap_num_blocks_t = nsamp/swap_divisions_in_t;
	int swap_num_blocks_f = nchans/swap_divisions_in_f;
	
	//printf("\nswap_divisions_in_t:%d\tswap_divisions_in_f:%d", CT, CF);
	//printf("\nswap_num_blocks_t:%d\tswap_num_blocks_f:%d",swap_num_blocks_t, swap_num_blocks_f);
	//printf("\nDIVISOR_t: %f\t DIVISOR_f: %f\n", (float) nsamp/swap_divisions_in_t, (float) nchans/swap_divisions_in_f);

	dim3 swap_threads_per_block(swap_divisions_in_t, swap_divisions_in_f);
	dim3 swap_num_blocks(swap_num_blocks_t, swap_num_blocks_f);

	hipDeviceSynchronize();
	swap<<<swap_num_blocks, swap_threads_per_block>>>(d_input, d_output, nchans, nsamp);
	hipDeviceSynchronize();

	timer.Stop();
	double time = (timer.Elapsed())/1000.0;
	
	//printf("\nPerformed Bin: %f (GPU estimate)", time);
	//printf("\nGops based on %.2f ops per channel per tsamp: %f",14.0,((15.0*(divisions_in_t*divisions_in_f*num_blocks_t*num_blocks_f))/(time))/1000000000.0);
	//printf("\nBN Device memory bandwidth in GB/s: %f", (2*(sizeof(float)+sizeof(unsigned short))*(divisions_in_t*divisions_in_f*num_blocks_t*num_blocks_f))/(time)/1000000000.0);

	hipMemset(d_output, 0, ((size_t) nchans)*((size_t) nsamp)*sizeof(float));
	//hipMemcpy(input_buffer, bin_buffer, sizeof(float)*nchans*(nsamp/2), hipMemcpyDeviceToDevice);
	//	getLastCudaError("Kernel execution failed");
}

